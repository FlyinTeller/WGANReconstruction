#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
//#include <torch/torch.h>
//#include <ATen/ATen.h>


#include <ATen/native/cuda/UpSample.cuh>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/KernelUtils.h>
#include <C10/macros/Macros.h>
#include <C10/cuda/CUDAException.h>
#include <ATen/cuda/detail/TensorInfo.cuh>

#include "CustomOperatorsBackend.cuh"

namespace at {
    namespace native {
        using at::native::GridSamplerInterpolation;
        using at::native::GridSamplerPadding;
        using namespace at::cuda::detail;

        //using at::native::detail::GridSamplerInterpolation;
        //using at::native::detail::GridSamplerPadding;

        namespace MyOperator {
            const int MY_CUDA_MAX_THREADS = 256;
            inline int MY_CUDA_GET_BLOCKS(const int64_t N) {
                AT_ASSERTM(N > 0, "CUDA kernel launch blocks must be positive, but got N=", N);
                constexpr int64_t max_int = std::numeric_limits<int>::max();

                // Round up division for positive number that cannot cause integer overflow
                auto block_num = (N - 1) / MY_CUDA_MAX_THREADS + 1;
                AT_ASSERTM(block_num <= max_int, "Can't schedule too many blocks on CUDA device");

                return static_cast<int>(block_num);
            }
            // Unnormalizes a coordinate from the -1 to +1 scale to its pixel index value,
            // where we view each pixel as an area between (idx - 0.5) and (idx + 0.5).
            // if align_corners: -1 and +1 get sent to the centers of the corner pixels
            //     -1 --> 0
            //     +1 --> (size - 1)
            //     scale_factor = (size - 1) / 2
            // if not align_corners: -1 and +1 get sent to the image edges
            //     -1 --> -0.5
            //     +1 --> (size - 1) + 0.5 == size - 0.5
            //     scale_factor = size / 2
            template <typename scalar_t>
            static __forceinline__ __device__
                scalar_t atoms_to_grid_unnormalize(scalar_t coord, int size, bool align_corners) {
                if (align_corners) {
                    // unnormalize coord from [-1, 1] to [0, size - 1]
                    return ((coord + 1.f) / 2) * (size - 1);
                }
                else {
                    // unnormalize coord from [-1, 1] to [-0.5, size - 0.5]
                    return ((coord + 1.f) * size - 1) / 2;
                }
            }

            // atoms_to_grid_unnormalize_set_grad works the same as atoms_to_grid_unnormalize
            // except that it also returns the `d output / d input` via pointer argument
            // `grad_in`.
            // This is useful in the backward pass of atoms_to_grid.
            template <typename scalar_t>
            static __forceinline__ __device__
                scalar_t atoms_to_grid_unnormalize_set_grad(scalar_t coord, int size,
                    bool align_corners, scalar_t* grad_in) {
                if (align_corners) {
                    // unnormalize coord from [-1, 1] to [0, size - 1]
                    *grad_in = static_cast<scalar_t>(size - 1) / 2;
                    return ((coord + 1.f) / 2) * (size - 1);
                }
                else {
                    // unnormalize coord from [-1, 1] to [-0.5, size - 0.5]
                    *grad_in = static_cast<scalar_t>(size) / 2;
                    return ((coord + 1.f) * size - 1) / 2;
                }
            }

            // Clips coordinates to between 0 and clip_limit - 1
            template <typename scalar_t>
            static __forceinline__ __device__
                scalar_t clip_coordinates(scalar_t in, int clip_limit) {
                return ::min(static_cast<scalar_t>(clip_limit - 1), ::max(in, static_cast<scalar_t>(0)));
            }

            // clip_coordinates_set_grad works similarly to clip_coordinates except that
            // it also returns the `d output / d input` via pointer argument `grad_in`.
            // This is useful in the backward pass of atoms_to_grid.
            template <typename scalar_t>
            static __forceinline__ __device__
                scalar_t clip_coordinates_set_grad(scalar_t in, int clip_limit, scalar_t* grad_in) {
                // Note that it is important for the gradient calculation that borders
                // are considered out of bounds.
                if (in <= static_cast<scalar_t>(0)) {
                    *grad_in = static_cast<scalar_t>(0);
                    return static_cast<scalar_t>(0);
                }
                else {
                    scalar_t max = static_cast<scalar_t>(clip_limit - 1);
                    if (in >= max) {
                        *grad_in = static_cast<scalar_t>(0);
                        return max;
                    }
                    else {
                        *grad_in = static_cast<scalar_t>(1);
                        return in;
                    }
                }
            }

            // Reflects coordinates until they fall between low and high (inclusive).
            // The bounds are passed as twice their value so that half-integer values
            // can be represented as ints.
            template <typename scalar_t>
            static __forceinline__ __device__
                scalar_t reflect_coordinates(scalar_t in, int twice_low, int twice_high) {
                if (twice_low == twice_high) {
                    return static_cast<scalar_t>(0);
                }
                scalar_t min = static_cast<scalar_t>(twice_low) / 2;
                scalar_t span = static_cast<scalar_t>(twice_high - twice_low) / 2;
                in = ::fabs(in - min);
                // `fmod` returns same sign as `in`, which is positive after the `fabs` above.
                scalar_t extra = ::fmod(in, span);
                int flips = static_cast<int>(::floor(in / span));
                if (flips % 2 == 0) {
                    return extra + min;
                }
                else {
                    return span - extra + min;
                }
            }

            // reflect_coordinates_set_grad works similarly to reflect_coordinates except
            // that it also returns the `d output / d input` via pointer argument
            // `grad_in`.
            // This is useful in the backward pass of atoms_to_grid.
            template <typename scalar_t>
            static __forceinline__ __device__
                scalar_t reflect_coordinates_set_grad(scalar_t in, int twice_low, int twice_high,
                    scalar_t* grad_in) {
                if (twice_low == twice_high) {
                    *grad_in = static_cast<scalar_t>(0);
                    return static_cast<scalar_t>(0);
                }
                int grad_in_mult_;
                scalar_t min = static_cast<scalar_t>(twice_low) / 2;
                scalar_t span = static_cast<scalar_t>(twice_high - twice_low) / 2;
                in = in - min;
                if (in < static_cast<scalar_t>(0)) {
                    grad_in_mult_ = -1;
                    in = -in;
                }
                else {
                    grad_in_mult_ = 1;
                }
                // `fmod` returns same sign as `in`, which is positive after the `if` above.
                scalar_t extra = ::fmod(in, span);
                int flips = static_cast<int>(::floor(in / span));
                if (flips % 2 == 0) {
                    *grad_in = static_cast<scalar_t>(grad_in_mult_);
                    return extra + min;
                }
                else {
                    *grad_in = static_cast<scalar_t>(-grad_in_mult_);
                    return span - extra + min;
                }
            }

            template<typename scalar_t>
            static __forceinline__ __device__
                scalar_t safe_downgrade_to_int_range(scalar_t x) {
                // -100.0 does not have special meaning. This is just to make sure
                // it's not within_bounds_2d or within_bounds_3d, and does not cause
                // undefined behavior. See #35506.
                if (x > INT_MAX - 1 || x < INT_MIN || !::isfinite(static_cast<double>(x)))
                    return static_cast<scalar_t>(-100.0);
                return x;
            }

            template<typename scalar_t>
            static __forceinline__ __device__
                scalar_t compute_coordinates(scalar_t coord, int size,
                    GridSamplerPadding padding_mode,
                    bool align_corners) {
                if (padding_mode == GridSamplerPadding::Border) {
                    // clip coordinates to image borders
                    coord = clip_coordinates(coord, size);
                }
                else if (padding_mode == GridSamplerPadding::Reflection) {
                    // reflect coordinates by image borders
                    if (align_corners) {
                        coord = reflect_coordinates(coord, 0, 2 * (size - 1));
                    }
                    else {
                        coord = reflect_coordinates(coord, -1, 2 * size - 1);
                    }
                    // clip coordinates to image borders
                    coord = clip_coordinates(coord, size);
                }

                coord = safe_downgrade_to_int_range(coord);
                return coord;
            }

            // Computes the pixel source index value for a grid coordinate
            template <typename scalar_t>
            static __forceinline__ __device__
                scalar_t atoms_to_grid_compute_source_index(
                    scalar_t coord,
                    int size,
                    GridSamplerPadding padding_mode,
                    bool align_corners) {
                coord = atoms_to_grid_unnormalize(coord, size, align_corners);
                coord = compute_coordinates(coord, size, padding_mode, align_corners);
                return coord;
            }

            // atoms_to_grid_compute_source_index_set_grad works similarly to
            // atoms_to_grid_compute_source_index except that it also returns the
            // `d output / d input` via pointer argument `grad_in`.
            // This is useful in the backward pass of atoms_to_grid.
            template <typename scalar_t>
            static __forceinline__ __device__
                scalar_t atoms_to_grid_compute_source_index_set_grad(
                    scalar_t coord,
                    int size,
                    GridSamplerPadding padding_mode,
                    bool align_corners,
                    scalar_t* grad_in) {
                scalar_t grad_clip, grad_refl;
                coord = atoms_to_grid_unnormalize_set_grad(coord, size, align_corners, grad_in);
                if (padding_mode == GridSamplerPadding::Border) {
                    // clip coordinates to image borders
                    coord = clip_coordinates_set_grad(coord, size, &grad_clip);
                    *grad_in = (*grad_in) * grad_clip;
                }
                else if (padding_mode == GridSamplerPadding::Reflection) {
                    // reflect coordinates by image borders
                    if (align_corners) {
                        coord = reflect_coordinates_set_grad(coord, 0, 2 * (size - 1), &grad_refl);
                    }
                    else {
                        coord = reflect_coordinates_set_grad(coord, -1, 2 * size - 1, &grad_refl);
                    }
                    // clip coordinates to image borders
                    coord = clip_coordinates_set_grad(coord, size, &grad_clip);
                    *grad_in = (*grad_in) * grad_refl * grad_clip;
                }

                coord = safe_downgrade_to_int_range(coord);
                return coord;
            }

            static __forceinline__ __device__
                bool within_bounds_2d(int h, int w, int H, int W) {
                return h >= 0 && h < H&& w >= 0 && w < W;
            }

            static __forceinline__ __device__
                bool within_bounds_3d(int d, int h, int w, int D, int H, int W) {
                return d >= 0 && d < D&& h >= 0 && h < H&& w >= 0 && w < W;
            }


            template<typename scalar_t>
            static __forceinline__ __device__
                void safe_add_2d(scalar_t* data, int h, int w,
                    int sH, int sW, int H, int W,
                    scalar_t delta) {
                if (within_bounds_2d(h, w, H, W)) {
                    gpuAtomicAdd(data + h * sH + w * sW, delta);
                }
            }

            template<typename scalar_t>
            static __forceinline__ __device__
                void safe_add_3d(scalar_t* data, int d, int h, int w,
                    int sD, int sH, int sW, int D, int H, int W,
                    scalar_t delta) {
                if (within_bounds_3d(d, h, w, D, H, W)) {
                    gpuAtomicAdd(data + d * sD + h * sH + w * sW, delta);
                }
            }

            template <typename scalar_t, typename index_t>
            static __forceinline__ __device__ scalar_t unsafe_access(scalar_t* data, index_t d, index_t h, index_t w, index_t sD, index_t sH, index_t sW) {
                return *(data + d * sD + h * sH + w * sW);
            }

            template <typename scalar_t, typename index_t>
            static __forceinline__ __device__ scalar_t unsafe_access(scalar_t* data, index_t h, index_t w, index_t sH, index_t sW) {
                return *(data + h * sH + w * sW);
            }

            template <typename scalar_t, typename index_t>
            __global__ void atoms_to_grid_3d_backward_kernel(
                const index_t nthreads,
                TensorInfo<scalar_t, index_t> grad_output,
                TensorInfo<scalar_t, index_t> intensities,
                TensorInfo<scalar_t, index_t> positions,
                TensorInfo<scalar_t, index_t> orientations,
                TensorInfo<scalar_t, index_t> shift,
                TensorInfo<scalar_t, index_t> grad_intensities,  // initialized to zeros
                TensorInfo<scalar_t, index_t> grad_positions,   // initialized to empty
                TensorInfo<scalar_t, index_t> grad_orientations,
                const GridSamplerInterpolation interpolation_mode,
                const GridSamplerPadding padding_mode,
                bool align_corners) {

                index_t int_D = intensities.sizes[1];
                index_t int_H = intensities.sizes[2];
                index_t int_W = intensities.sizes[3];
                index_t gOut_D = grad_output.sizes[1];
                index_t gOut_H = grad_output.sizes[2];
                index_t gOut_W = grad_output.sizes[3];
                index_t int_sN = intensities.strides[0];
                index_t int_sW = intensities.strides[1];
                index_t pos_sN = positions.strides[0];
                index_t pos_sW = positions.strides[1];
                index_t pos_sCoor = positions.strides[2];
                index_t orr_sN = orientations.strides[0];
                index_t orr_sR = orientations.strides[1];
                index_t orr_sC = orientations.strides[2];
                index_t shift_sN = shift.strides[0];
                index_t shift_sCoor = shift.strides[1];
                index_t gOut_sN = grad_output.strides[0];
                index_t gOut_sD = grad_output.strides[1];
                index_t gOut_sH = grad_output.strides[2];
                index_t gOut_sW = grad_output.strides[3];
                index_t gInp_sN = grad_intensities.strides[0];
                index_t gInp_sW = grad_intensities.strides[1];

                CUDA_KERNEL_LOOP_TYPE(index, nthreads, index_t) {
                    const index_t w = index % int_W;
                    const index_t n = index / (int_W);
                    const index_t pos_offset = n * pos_sN + w * pos_sW;

                    // get the corresponding input x, y, z co-ordinates from pos
                    scalar_t ix = positions.data[pos_offset] + shift.data[n * shift_sN];
                    scalar_t iy = positions.data[pos_offset + pos_sCoor] + shift.data[n * shift_sN + shift_sCoor];
                    scalar_t iz = positions.data[pos_offset + 2 * pos_sCoor] + shift.data[n * shift_sN + 2 * shift_sCoor];

                    //rotate coordinates
                    scalar_t ox, oy, oz;
                    ox = ix;
                    oy = iy;
                    oz = iz;
                    scalar_t tIx = ix;
                    scalar_t tIy = iy;
                    scalar_t tIz = iz;
                    matMult(orientations.data + n * orr_sN, orr_sC, orr_sR, tIx, tIy, tIz, &ix, &iy, &iz);

                    scalar_t gix_mult, giy_mult, giz_mult;
                    ix = atoms_to_grid_compute_source_index_set_grad(ix, gOut_W, padding_mode, align_corners, &gix_mult);
                    iy = atoms_to_grid_compute_source_index_set_grad(iy, gOut_H, padding_mode, align_corners, &giy_mult);
                    iz = atoms_to_grid_compute_source_index_set_grad(iz, gOut_D, padding_mode, align_corners, &giz_mult);

                    if (interpolation_mode == GridSamplerInterpolation::Bilinear) {
                        // get corner pixel values from (x, y, z)
                        // for 4d, we used north-east-south-west
                        // for 5d, we add top-bottom
                        index_t ix_tnw = static_cast<index_t>((ix));
                        index_t iy_tnw = static_cast<index_t>((iy));
                        index_t iz_tnw = static_cast<index_t>((iz));

                        index_t ix_tne = ix_tnw + 1;
                        index_t iy_tne = iy_tnw;
                        index_t iz_tne = iz_tnw;

                        index_t ix_tsw = ix_tnw;
                        index_t iy_tsw = iy_tnw + 1;
                        index_t iz_tsw = iz_tnw;

                        index_t ix_tse = ix_tnw + 1;
                        index_t iy_tse = iy_tnw + 1;
                        index_t iz_tse = iz_tnw;

                        index_t ix_bnw = ix_tnw;
                        index_t iy_bnw = iy_tnw;
                        index_t iz_bnw = iz_tnw + 1;

                        index_t ix_bne = ix_tnw + 1;
                        index_t iy_bne = iy_tnw;
                        index_t iz_bne = iz_tnw + 1;

                        index_t ix_bsw = ix_tnw;
                        index_t iy_bsw = iy_tnw + 1;
                        index_t iz_bsw = iz_tnw + 1;

                        index_t ix_bse = ix_tnw + 1;
                        index_t iy_bse = iy_tnw + 1;
                        index_t iz_bse = iz_tnw + 1;

                        // get surfaces to each neighbor:
                        scalar_t tnw = (ix_bse - ix) * (iy_bse - iy) * (iz_bse - iz);
                        scalar_t tne = (ix - ix_bsw) * (iy_bsw - iy) * (iz_bsw - iz);
                        scalar_t tsw = (ix_bne - ix) * (iy - iy_bne) * (iz_bne - iz);
                        scalar_t tse = (ix - ix_bnw) * (iy - iy_bnw) * (iz_bnw - iz);
                        scalar_t bnw = (ix_tse - ix) * (iy_tse - iy) * (iz - iz_tse);
                        scalar_t bne = (ix - ix_tsw) * (iy_tsw - iy) * (iz - iz_tsw);
                        scalar_t bsw = (ix_tne - ix) * (iy - iy_tne) * (iz - iz_tne);
                        scalar_t bse = (ix - ix_tnw) * (iy - iy_tnw) * (iz - iz_tnw);

                        scalar_t gix = static_cast<scalar_t>(0), giy = static_cast<scalar_t>(0), giz = static_cast<scalar_t>(0), gi = static_cast<scalar_t>(0);
                        auto int_ptr_NC = intensities.data + n * int_sN;
                        auto gInp_ptr_NCDHW = grad_intensities.data + n * gInp_sN + w * gInp_sW;
                        auto int_val_NCDHW = *(intensities.data + n * int_sN + w * int_sW);
                        auto gOut_ptr_NC = grad_output.data + n * gOut_sN;
                        // calculate grad_pos
                        if (within_bounds_3d(iz_tnw, iy_tnw, ix_tnw, gOut_D, gOut_H, gOut_W)) {
                            auto gOut = unsafe_access(gOut_ptr_NC, iz_tnw, iy_tnw, ix_tnw, gOut_sD, gOut_sH, gOut_sW);
                            gi += tnw * gOut;
                            gix -= int_val_NCDHW * (iy_bse - iy) * (iz_bse - iz) * gOut;
                            giy -= int_val_NCDHW * (ix_bse - ix) * (iz_bse - iz) * gOut;
                            giz -= int_val_NCDHW * (ix_bse - ix) * (iy_bse - iy) * gOut;
                        }
                        if (within_bounds_3d(iz_tne, iy_tne, ix_tne, gOut_D, gOut_H, gOut_W)) {
                            auto gOut = unsafe_access(gOut_ptr_NC, iz_tne, iy_tne, ix_tne, gOut_sD, gOut_sH, gOut_sW);
                            gi += tne * gOut;
                            gix += int_val_NCDHW * (iy_bsw - iy) * (iz_bsw - iz) * gOut;
                            giy -= int_val_NCDHW * (ix - ix_bsw) * (iz_bsw - iz) * gOut;
                            giz -= int_val_NCDHW * (ix - ix_bsw) * (iy_bsw - iy) * gOut;
                        }
                        if (within_bounds_3d(iz_tsw, iy_tsw, ix_tsw, gOut_D, gOut_H, gOut_W)) {
                            auto gOut = unsafe_access(gOut_ptr_NC, iz_tsw, iy_tsw, ix_tsw, gOut_sD, gOut_sH, gOut_sW);
                            gi += tsw * gOut;
                            gix -= int_val_NCDHW * (iy - iy_bne) * (iz_bne - iz) * gOut;
                            giy += int_val_NCDHW * (ix_bne - ix) * (iz_bne - iz) * gOut;
                            giz -= int_val_NCDHW * (ix_bne - ix) * (iy - iy_bne) * gOut;
                        }
                        if (within_bounds_3d(iz_tse, iy_tse, ix_tse, gOut_D, gOut_H, gOut_W)) {
                            auto gOut = unsafe_access(gOut_ptr_NC, iz_tse, iy_tse, ix_tse, gOut_sD, gOut_sH, gOut_sW);
                            gi += tse * gOut;
                            gix += int_val_NCDHW * (iy - iy_bnw) * (iz_bnw - iz) * gOut;
                            giy += int_val_NCDHW * (ix - ix_bnw) * (iz_bnw - iz) * gOut;
                            giz -= int_val_NCDHW * (ix - ix_bnw) * (iy - iy_bnw) * gOut;
                        }
                        if (within_bounds_3d(iz_bnw, iy_bnw, ix_bnw, gOut_D, gOut_H, gOut_W)) {
                            auto gOut = unsafe_access(gOut_ptr_NC, iz_bnw, iy_bnw, ix_bnw, gOut_sD, gOut_sH, gOut_sW);
                            gi += bnw * gOut;
                            gix -= int_val_NCDHW * (iy_tse - iy) * (iz - iz_tse) * gOut;
                            giy -= int_val_NCDHW * (ix_tse - ix) * (iz - iz_tse) * gOut;
                            giz += int_val_NCDHW * (ix_tse - ix) * (iy_tse - iy) * gOut;
                        }
                        if (within_bounds_3d(iz_bne, iy_bne, ix_bne, gOut_D, gOut_H, gOut_W)) {
                            auto gOut = unsafe_access(gOut_ptr_NC, iz_bne, iy_bne, ix_bne, gOut_sD, gOut_sH, gOut_sW);
                            gi += bne * gOut;
                            gix += int_val_NCDHW * (iy_tsw - iy) * (iz - iz_tsw) * gOut;
                            giy -= int_val_NCDHW * (ix - ix_tsw) * (iz - iz_tsw) * gOut;
                            giz += int_val_NCDHW * (ix - ix_tsw) * (iy_tsw - iy) * gOut;
                        }
                        if (within_bounds_3d(iz_bsw, iy_bsw, ix_bsw, gOut_D, gOut_H, gOut_W)) {
                            auto gOut = unsafe_access(gOut_ptr_NC, iz_bsw, iy_bsw, ix_bsw, gOut_sD, gOut_sH, gOut_sW);
                            gi += bsw * gOut;
                            gix -= int_val_NCDHW * (iy - iy_tne) * (iz - iz_tne) * gOut;
                            giy += int_val_NCDHW * (ix_tne - ix) * (iz - iz_tne) * gOut;
                            giz += int_val_NCDHW * (ix_tne - ix) * (iy - iy_tne) * gOut;
                        }
                        if (within_bounds_3d(iz_bse, iy_bse, ix_bse, gOut_D, gOut_H, gOut_W)) {
                            auto gOut = unsafe_access(gOut_ptr_NC, iz_bse, iy_bse, ix_bse, gOut_sD, gOut_sH, gOut_sW);
                            gi += bse * gOut;
                            gix += int_val_NCDHW * (iy - iy_tnw) * (iz - iz_tnw) * gOut;
                            giy += int_val_NCDHW * (ix - ix_tnw) * (iz - iz_tnw) * gOut;
                            giz += int_val_NCDHW * (ix - ix_tnw) * (iy - iy_tnw) * gOut;
                        }
                        *(gInp_ptr_NCDHW) = gi;

                        tIx = gix * gix_mult;
                        tIy = giy * giy_mult;
                        tIz = giz * giz_mult;
                        safe_matGrad(grad_orientations.data + n * orr_sN, orr_sC, orr_sR, ox, oy, oz, tIx, tIy, tIz);

                        matMultT(orientations.data + n * orr_sN, orr_sC, orr_sR, tIx, tIy, tIz, &gix, &giy, &giz);
                        grad_positions.data[pos_offset] = gix;
                        grad_positions.data[pos_offset + pos_sCoor] = giy;
                        grad_positions.data[pos_offset + 2 * pos_sCoor] = giz;
                    }
                }
            }


            template <typename scalar_t, typename index_t>
            __global__ void atoms_to_grid_3d_kernel(
                const index_t nthreads,
                TensorInfo<scalar_t, index_t> intensities,
                TensorInfo<scalar_t, index_t> positions,
                TensorInfo<scalar_t, index_t> orientations,
                TensorInfo<scalar_t, index_t> shift,
                TensorInfo<scalar_t, index_t> output, //Initialized to zeros
                const GridSamplerInterpolation interpolation_mode,
                const GridSamplerPadding padding_mode,
                bool align_corners) {


                index_t out_D = output.sizes[1];
                index_t out_H = output.sizes[2];
                index_t out_W = output.sizes[3];
                index_t pos_N = positions.sizes[0];
                index_t pos_W = positions.sizes[1];
                index_t orr_sN = orientations.strides[0];
                index_t orr_sR = orientations.strides[1];
                index_t orr_sC = orientations.strides[2];
                index_t out_sN = output.strides[0];
                index_t out_sD = output.strides[1];
                index_t out_sH = output.strides[2];
                index_t out_sW = output.strides[3];
                index_t pos_sN = positions.strides[0];
                index_t pos_sW = positions.strides[1];
                index_t pos_sCoor = positions.strides[2];
                index_t shift_sN = shift.strides[0];
                index_t shift_sCoor = shift.strides[1];
                index_t int_sN = intensities.strides[0];
                index_t int_sW = intensities.strides[1];

                CUDA_KERNEL_LOOP_TYPE(index, nthreads, index_t) {
                    const index_t w = index % pos_W;
                    const index_t n = index / (pos_W);
                    const auto pos_offset = n * pos_sN + w * pos_sW;

                    // get the corresponding output x, y, z co-ordinates from pos
                    // 1st the coordinates saved in pos, i.e. in [-1, 1]
                    scalar_t ix = positions.data[pos_offset] + shift.data[n * shift_sN];
                    scalar_t iy = positions.data[pos_offset + pos_sCoor] + shift.data[n * shift_sN + shift_sCoor];
                    scalar_t iz = positions.data[pos_offset + 2 * pos_sCoor] + shift.data[n * shift_sN + 2 * shift_sCoor];

                    scalar_t tIx = ix;
                    scalar_t tIy = iy;
                    scalar_t tIz = iz;
                    matMult(orientations.data + n * orr_sN, orr_sC, orr_sR, tIx, tIy, tIz, &ix, &iy, &iz);

                    // 2nd, unnormalized coordinates in [0, outsize-1]
                    ix = atoms_to_grid_compute_source_index(ix, out_W, padding_mode, align_corners);
                    iy = atoms_to_grid_compute_source_index(iy, out_H, padding_mode, align_corners);
                    iz = atoms_to_grid_compute_source_index(iz, out_D, padding_mode, align_corners);

                    // get corner pixel values from (x, y, z)
                    /*index_t ix_tnw = static_cast<index_t>((ix));
                    index_t iy_tnw = static_cast<index_t>((iy));
                    index_t iz_tnw = static_cast<index_t>((iz));*/
                    index_t ix_tnw = static_cast<index_t>((ix));
                    index_t iy_tnw = static_cast<index_t>((iy));
                    index_t iz_tnw = static_cast<index_t>((iz));
                    index_t ix_tne = ix_tnw + 1;
                    index_t iy_tne = iy_tnw;
                    index_t iz_tne = iz_tnw;

                    index_t ix_tsw = ix_tnw;
                    index_t iy_tsw = iy_tnw + 1;
                    index_t iz_tsw = iz_tnw;

                    index_t ix_tse = ix_tnw + 1;
                    index_t iy_tse = iy_tnw + 1;
                    index_t iz_tse = iz_tnw;

                    index_t ix_bnw = ix_tnw;
                    index_t iy_bnw = iy_tnw;
                    index_t iz_bnw = iz_tnw + 1;

                    index_t ix_bne = ix_tnw + 1;
                    index_t iy_bne = iy_tnw;
                    index_t iz_bne = iz_tnw + 1;

                    index_t ix_bsw = ix_tnw;
                    index_t iy_bsw = iy_tnw + 1;
                    index_t iz_bsw = iz_tnw + 1;

                    index_t ix_bse = ix_tnw + 1;
                    index_t iy_bse = iy_tnw + 1;
                    index_t iz_bse = iz_tnw + 1;

                    // get surfaces to each neighbor on cartesian grid:
                    scalar_t tnw = (ix_bse - ix) * (iy_bse - iy) * (iz_bse - iz);
                    scalar_t tne = (ix - ix_bsw) * (iy_bsw - iy) * (iz_bsw - iz);
                    scalar_t tsw = (ix_bne - ix) * (iy - iy_bne) * (iz_bne - iz);
                    scalar_t tse = (ix - ix_bnw) * (iy - iy_bnw) * (iz_bnw - iz);
                    scalar_t bnw = (ix_tse - ix) * (iy_tse - iy) * (iz - iz_tse);
                    scalar_t bne = (ix - ix_tsw) * (iy_tsw - iy) * (iz - iz_tsw);
                    scalar_t bsw = (ix_tne - ix) * (iy - iy_tne) * (iz - iz_tne);
                    scalar_t bse = (ix - ix_tnw) * (iy - iy_tnw) * (iz - iz_tnw);

                    //intensity value at current grid position
                    scalar_t int_val_NCDHW = *(intensities.data + n * int_sN + w * int_sW);
                    scalar_t* out_ptr_NC = output.data + n * out_sN;

                    // calculate bilinear weighted pixel value and set output pixel
                    safe_add_3d(out_ptr_NC, iz_tnw, iy_tnw, ix_tnw, out_sD, out_sH, out_sW, out_D, out_H, out_W, int_val_NCDHW * tnw);
                    safe_add_3d(out_ptr_NC, iz_tne, iy_tne, ix_tne, out_sD, out_sH, out_sW, out_D, out_H, out_W, int_val_NCDHW * tne);
                    safe_add_3d(out_ptr_NC, iz_tsw, iy_tsw, ix_tsw, out_sD, out_sH, out_sW, out_D, out_H, out_W, int_val_NCDHW * tsw);
                    safe_add_3d(out_ptr_NC, iz_tse, iy_tse, ix_tse, out_sD, out_sH, out_sW, out_D, out_H, out_W, int_val_NCDHW * tse);
                    safe_add_3d(out_ptr_NC, iz_bnw, iy_bnw, ix_bnw, out_sD, out_sH, out_sW, out_D, out_H, out_W, int_val_NCDHW * bnw);
                    safe_add_3d(out_ptr_NC, iz_bne, iy_bne, ix_bne, out_sD, out_sH, out_sW, out_D, out_H, out_W, int_val_NCDHW * bne);
                    safe_add_3d(out_ptr_NC, iz_bsw, iy_bsw, ix_bsw, out_sD, out_sH, out_sW, out_D, out_H, out_W, int_val_NCDHW * bsw);
                    safe_add_3d(out_ptr_NC, iz_bse, iy_bse, ix_bse, out_sD, out_sH, out_sW, out_D, out_H, out_W, int_val_NCDHW * bse);
                }
            }


            template <typename scalar_t, typename index_t>
            static  __forceinline__ __device__ void matMult(scalar_t* mat, index_t sC, index_t sR, scalar_t x, scalar_t y, scalar_t z, scalar_t* rX, scalar_t* rY, scalar_t* rZ) {
                *(rX) = x * (*mat) + y * (*(mat + sC)) + z * (*(mat + 2 * sC));
                mat += sR;
                *(rY) = x * (*mat) + y * (*(mat + sC)) + z * (*(mat + 2 * sC));
                mat += sR;
                *(rZ) = x * (*mat) + y * (*(mat + sC)) + z * (*(mat + 2 * sC));
            }

            template <typename scalar_t, typename index_t>
            static  __forceinline__ __device__ void safe_matGrad(scalar_t* grad_mat, index_t sC, index_t sR, scalar_t x, scalar_t y, scalar_t z, scalar_t gx, scalar_t gy, scalar_t gz) {
                gpuAtomicAdd(grad_mat, x * gx);
                gpuAtomicAdd(grad_mat + sC, y * gx);
                gpuAtomicAdd(grad_mat + 2 * sC, z * gx);
                grad_mat += sR;
                gpuAtomicAdd(grad_mat, x * gy);
                gpuAtomicAdd(grad_mat + sC, y * gy);
                gpuAtomicAdd(grad_mat + 2 * sC, z * gy);
                grad_mat += sR;
                gpuAtomicAdd(grad_mat, x * gz);
                gpuAtomicAdd(grad_mat + sC, y * gz);
                gpuAtomicAdd(grad_mat + 2 * sC, z * gz);
            }

            template <typename scalar_t, typename index_t>
            static  __forceinline__ __device__ void matMultT(scalar_t* mat, index_t sC, index_t sR, scalar_t x, scalar_t y, scalar_t z, scalar_t* rX, scalar_t* rY, scalar_t* rZ) {
                *(rX) = x * (*mat) + y * (*(mat + sR)) + z * (*(mat + 2 * sR));
                mat += sC;
                *(rY) = x * (*mat) + y * (*(mat + sR)) + z * (*(mat + 2 * sR));
                mat += sC;
                *(rZ) = x * (*mat) + y * (*(mat + sR)) + z * (*(mat + 2 * sR));
            }



            template <typename scalar_t, typename index_t>
            __global__ void projectAtoms_backwards_kernel(
                const index_t nthreads,
                TensorInfo<scalar_t, index_t> positions,
                TensorInfo<scalar_t, index_t> intensities,
                TensorInfo<scalar_t, index_t> orientation,
                TensorInfo<scalar_t, index_t> shift,
                TensorInfo<scalar_t, index_t> grad_output,
                TensorInfo<scalar_t, index_t> grad_positions,
                TensorInfo<scalar_t, index_t> grad_intensities,
                TensorInfo<scalar_t, index_t> grad_orientations,
                int64_t x, int64_t y, int64_t z)
            {


                index_t pos_N = positions.sizes[0];
                index_t pos_W = positions.sizes[1];
                index_t gOut_H = grad_output.sizes[1];
                index_t gOut_W = grad_output.sizes[2];

                index_t ints_sN = intensities.strides[0];
                index_t ints_sW = intensities.strides[1];
                index_t orr_sN = orientation.strides[0];
                index_t orr_sR = orientation.strides[1];
                index_t orr_sC = orientation.strides[2];
                index_t pos_sN = positions.strides[0];
                index_t pos_sW = positions.strides[1];
                index_t pos_sCoor = positions.strides[2];
                index_t shift_sN = shift.strides[0];
                index_t shift_sCoor = shift.strides[1];

                index_t gOut_sN = grad_output.strides[0];
                index_t gOut_sH = grad_output.strides[1];
                index_t gOut_sW = grad_output.strides[2];
                index_t gInts_sN = grad_intensities.strides[0];
                index_t gInts_sW = grad_intensities.strides[1];
                index_t gPos_sN = grad_positions.strides[0];
                index_t gPos_sW = grad_positions.strides[1];
                index_t gPos_sCoor = grad_positions.strides[2];



                CUDA_KERNEL_LOOP_TYPE(index, nthreads, index_t) {
                    const index_t w = index % pos_W;
                    const index_t n = index / (pos_W);
                    const auto pos_offset = n * pos_sN + w * pos_sW;
                    // get the corresponding output x, y, z co-ordinates from grid
                    // 1st the coordinates saved in grid, i.e. in [-1, 1]
                    scalar_t ix = positions.data[pos_offset] + shift.data[n * shift_sN];
                    scalar_t iy = positions.data[pos_offset + pos_sCoor] + shift.data[n * shift_sN + shift_sCoor];
                    scalar_t iz = positions.data[pos_offset + 2 * pos_sCoor] + shift.data[n * shift_sN + 2 * shift_sCoor];
                    scalar_t ox, oy, oz;
                    ox = ix;
                    oy = iy;
                    oz = iz;
                    //rotate coordinates
                    scalar_t tIx = ix;
                    scalar_t tIy = iy;
                    scalar_t tIz = iz;
                    matMult(orientation.data + n * orr_sN, orr_sC, orr_sR, tIx, tIy, tIz, &ix, &iy, &iz);

                    // 2nd, unnormalized coordinates in [0, outsize-1]
                    scalar_t gix_mult, giy_mult, giz_mult;
                    ix = atoms_to_grid_compute_source_index_set_grad(ix, x, GridSamplerPadding::Zeros, true, &gix_mult);
                    iy = atoms_to_grid_compute_source_index_set_grad(iy, y, GridSamplerPadding::Zeros, true, &giy_mult);
                    iz = atoms_to_grid_compute_source_index_set_grad(iz, z, GridSamplerPadding::Zeros, true, &giz_mult);


                    if (!within_bounds_2d(ix, iy, x, y)) {
                        continue;
                    }
                    if (true) {//interpolation_mode == GridSamplerInterpolation::Bilinear) {
                        // get corner pixel values from (x, y, z)
                        index_t ix_nw = static_cast<index_t>((ix));
                        index_t iy_nw = static_cast<index_t>((iy));

                        index_t ix_ne = ix_nw + 1;
                        index_t iy_ne = iy_nw;

                        index_t ix_sw = ix_nw;
                        index_t iy_sw = iy_nw + 1;

                        index_t ix_se = ix_nw + 1;
                        index_t iy_se = iy_nw + 1;

                        // get surfaces to each neighbor on cartesian grid:
                        scalar_t nw = (ix_se - ix) * (iy_se - iy);
                        scalar_t ne = (ix - ix_sw) * (iy_sw - iy);
                        scalar_t sw = (ix_ne - ix) * (iy - iy_ne);
                        scalar_t se = (ix - ix_nw) * (iy - iy_nw);

                        //intensity value at current grid position
                        scalar_t gix = static_cast<scalar_t>(0), giy = static_cast<scalar_t>(0), giz = static_cast<scalar_t>(0), gi = static_cast<scalar_t>(0);
                        auto gInp_ptr_NCDHW = grad_intensities.data + n * gInts_sN + w * gInts_sW;
                        auto ints_val_NCDHW = *(intensities.data + n * ints_sN + w * ints_sW);
                        auto gOut_ptr_NC = grad_output.data + n * gOut_sN;
                        // calculate grad_grid
                        if (within_bounds_2d(iy_nw, ix_nw, gOut_H, gOut_W)) {
                            auto gOut = unsafe_access(gOut_ptr_NC, iy_nw, ix_nw, gOut_sH, gOut_sW);
                            gi += nw * gOut;
                            gix -= ints_val_NCDHW * (iy_se - iy) * gOut;
                            giy -= ints_val_NCDHW * (ix_se - ix) * gOut;
                        }
                        if (within_bounds_2d(iy_ne, ix_ne, gOut_H, gOut_W)) {
                            auto gOut = unsafe_access(gOut_ptr_NC, iy_ne, ix_ne, gOut_sH, gOut_sW);
                            gi += ne * gOut;
                            gix += ints_val_NCDHW * (iy_sw - iy) * gOut;
                            giy -= ints_val_NCDHW * (ix - ix_sw) * gOut;
                        }
                        if (within_bounds_2d(iy_sw, ix_sw, gOut_H, gOut_W)) {
                            auto gOut = unsafe_access(gOut_ptr_NC, iy_sw, ix_sw, gOut_sH, gOut_sW);
                            gi += sw * gOut;
                            gix -= ints_val_NCDHW * (iy - iy_ne) * gOut;
                            giy += ints_val_NCDHW * (ix_ne - ix) * gOut;
                        }
                        if (within_bounds_2d(iy_se, ix_se, gOut_H, gOut_W)) {
                            auto gOut = unsafe_access(gOut_ptr_NC, iy_se, ix_se, gOut_sH, gOut_sW);
                            gi += se * gOut;
                            gix += ints_val_NCDHW * (iy - iy_nw) * gOut;
                            giy += ints_val_NCDHW * (ix - ix_nw) * gOut;
                        }

                        *(gInp_ptr_NCDHW) = gi;
                        tIx = gix * gix_mult;
                        tIy = giy * giy_mult;
                        tIz = giz * giz_mult;
                        safe_matGrad(grad_orientations.data + n * orr_sN, orr_sC, orr_sR, ox, oy, oz, tIx, tIy, tIz);

                        matMultT(orientation.data + n * orr_sN, orr_sC, orr_sR, tIx, tIy, tIz, &gix, &giy, &giz);
                        grad_positions.data[pos_offset] = gix;
                        grad_positions.data[pos_offset + pos_sCoor] = giy;
                        grad_positions.data[pos_offset + 2 * pos_sCoor] = giz;
                    }

                }
            }

            template <typename scalar_t, typename index_t>
            __global__ void projectAtoms_kernel(
                const index_t nthreads,
                TensorInfo<scalar_t, index_t> positions,
                TensorInfo<scalar_t, index_t> intensities,
                TensorInfo<scalar_t, index_t> orientation,
                TensorInfo<scalar_t, index_t> shift,
                TensorInfo<scalar_t, index_t> output,
                int64_t x, int64_t y, int64_t z)
            {

                index_t out_H = output.sizes[1];
                index_t out_W = output.sizes[2];
                index_t pos_N = positions.sizes[0];
                index_t pos_W = positions.sizes[1];


                index_t ints_sN = intensities.strides[0];
                index_t ints_sW = intensities.strides[1];
                index_t orr_sN = orientation.strides[0];
                index_t orr_sR = orientation.strides[1];
                index_t orr_sC = orientation.strides[2];
                index_t out_sN = output.strides[0];
                index_t out_sH = output.strides[1];
                index_t out_sW = output.strides[2];
                index_t pos_sN = positions.strides[0];
                index_t pos_sW = positions.strides[1];
                index_t pos_sCoor = positions.strides[2];
                index_t shift_sN = shift.strides[0];
                index_t shift_sCoor = shift.strides[1];

                CUDA_KERNEL_LOOP_TYPE(index, nthreads, index_t) {
                    const index_t w = index % pos_W;
                    const index_t n = index / (pos_W);
                    const auto pos_offset = n * pos_sN + w * pos_sW;

                    // get the corresponding output x, y, z co-ordinates from grid
                    // 1st the coordinates saved in grid, i.e. in [-1, 1]
                    scalar_t ix = positions.data[pos_offset] + shift.data[n * shift_sN];
                    scalar_t iy = positions.data[pos_offset + pos_sCoor] + shift.data[n * shift_sN + shift_sCoor];
                    scalar_t iz = positions.data[pos_offset + 2 * pos_sCoor] + shift.data[n * shift_sN + 2 * shift_sCoor];

                    // 2nd, unnormalized coordinates in [0, outsize-1]
                    ix = atoms_to_grid_compute_source_index(ix, x, GridSamplerPadding::Zeros, true);
                    iy = atoms_to_grid_compute_source_index(iy, y, GridSamplerPadding::Zeros, true);
                    iz = atoms_to_grid_compute_source_index(iz, z, GridSamplerPadding::Zeros, true);
                    scalar_t tIx = ix - x / 2;
                    scalar_t tIy = iy - y / 2;
                    scalar_t tIz = iz - z / 2;
                    matMult(orientation.data + n * orr_sN, orr_sC, orr_sR, tIx, tIy, tIz, &ix, &iy, &iz);
                    ix += x / 2;
                    iy += y / 2;
                    iz += z / 2;

                    // get corner pixel values from (x, y, z)
                    index_t ix_nw = static_cast<index_t>((ix));
                    index_t iy_nw = static_cast<index_t>((iy));

                    index_t ix_ne = ix_nw + 1;
                    index_t iy_ne = iy_nw;

                    index_t ix_sw = ix_nw;
                    index_t iy_sw = iy_nw + 1;

                    index_t ix_se = ix_nw + 1;
                    index_t iy_se = iy_nw + 1;

                    // get surfaces to each neighbor on cartesian grid:
                    scalar_t nw = (ix_se - ix) * (iy_se - iy);
                    scalar_t ne = (ix - ix_sw) * (iy_sw - iy);
                    scalar_t sw = (ix_ne - ix) * (iy - iy_ne);
                    scalar_t se = (ix - ix_nw) * (iy - iy_nw);

                    //intensity value at current grid position
                    scalar_t ints_val_NCHW = *(intensities.data + n * ints_sN + w * ints_sW);
                    scalar_t* out_ptr_NC = output.data + n * out_sN;

                    // calculate bilinear weighted pixel value and set output pixel
                    safe_add_2d(out_ptr_NC, iy_nw, ix_nw, out_sH, out_sW, out_H, out_W, ints_val_NCHW * nw);
                    safe_add_2d(out_ptr_NC, iy_ne, ix_ne, out_sH, out_sW, out_H, out_W, ints_val_NCHW * ne);
                    safe_add_2d(out_ptr_NC, iy_sw, ix_sw, out_sH, out_sW, out_H, out_W, ints_val_NCHW * sw);
                    safe_add_2d(out_ptr_NC, iy_se, ix_se, out_sH, out_sW, out_H, out_W, ints_val_NCHW * se);
                }
            }

            template <typename scalar_t, typename index_t>
            __global__ void grid_sampler_and_project_3d_kernel(
                const index_t nthreads,
                TensorInfo<scalar_t, index_t> input,
                TensorInfo<scalar_t, index_t> grid,
                TensorInfo<scalar_t, index_t> output,
                const GridSamplerInterpolation interpolation_mode,
                const GridSamplerPadding padding_mode,
                bool align_corners) {

                index_t C = input.sizes[1];
                index_t inp_D = input.sizes[2];
                index_t inp_H = input.sizes[3];
                index_t inp_W = input.sizes[4];
                index_t grid_D = grid.sizes[1];
                index_t out_H = grid.sizes[2];
                index_t out_W = grid.sizes[3];
                //index_t inp_sN = input.strides[0];
                index_t inp_sC = input.strides[1];
                index_t inp_sD = input.strides[2];
                index_t inp_sH = input.strides[3];
                index_t inp_sW = input.strides[4];
                index_t grid_sN = grid.strides[0];
                index_t grid_sD = grid.strides[1];
                index_t grid_sH = grid.strides[2];
                index_t grid_sW = grid.strides[3];
                index_t grid_sCoor = grid.strides[4];
                index_t out_sN = output.strides[0];
                index_t out_sC = output.strides[1];
                index_t out_sH = output.strides[2];
                index_t out_sW = output.strides[3];

                CUDA_KERNEL_LOOP_TYPE(index, nthreads, index_t) {
                    const index_t w = index % out_W;
                    const index_t h = (index / out_W) % out_H;
                    //const index_t d = (index / (out_H * out_W)) % out_D;

                    const index_t n = index / (out_H * out_W);

                    //This prevents atomic add operations further down
                    for (index_t d = 0; d < grid_D; d++) {
                        const index_t grid_offset = n * grid_sN + d * grid_sD + h * grid_sH + w * grid_sW;

                        // get the corresponding input x, y, z co-ordinates from grid
                        scalar_t ix = grid.data[grid_offset];
                        scalar_t iy = grid.data[grid_offset + grid_sCoor];
                        scalar_t iz = grid.data[grid_offset + 2 * grid_sCoor];

                        ix = atoms_to_grid_compute_source_index(ix, inp_W, padding_mode, align_corners);
                        iy = atoms_to_grid_compute_source_index(iy, inp_H, padding_mode, align_corners);
                        iz = atoms_to_grid_compute_source_index(iz, inp_D, padding_mode, align_corners);

                        if (interpolation_mode == GridSamplerInterpolation::Bilinear) {


                            // get corner pixel values from (x, y, z)
                            // for 4d, we used north-east-south-west
                            // for 5d, we add top-bottom
                            index_t ix_tnw = static_cast<index_t>(::floor(ix));
                            index_t iy_tnw = static_cast<index_t>(::floor(iy));
                            index_t iz_tnw = static_cast<index_t>(::floor(iz));

                            index_t ix_tne = ix_tnw + 1;
                            index_t iy_tne = iy_tnw;
                            index_t iz_tne = iz_tnw;

                            index_t ix_tsw = ix_tnw;
                            index_t iy_tsw = iy_tnw + 1;
                            index_t iz_tsw = iz_tnw;

                            index_t ix_tse = ix_tnw + 1;
                            index_t iy_tse = iy_tnw + 1;
                            index_t iz_tse = iz_tnw;

                            index_t ix_bnw = ix_tnw;
                            index_t iy_bnw = iy_tnw;
                            index_t iz_bnw = iz_tnw + 1;

                            index_t ix_bne = ix_tnw + 1;
                            index_t iy_bne = iy_tnw;
                            index_t iz_bne = iz_tnw + 1;

                            index_t ix_bsw = ix_tnw;
                            index_t iy_bsw = iy_tnw + 1;
                            index_t iz_bsw = iz_tnw + 1;

                            index_t ix_bse = ix_tnw + 1;
                            index_t iy_bse = iy_tnw + 1;
                            index_t iz_bse = iz_tnw + 1;

                            // get surfaces to each neighbor:
                            scalar_t tnw = (ix_bse - ix) * (iy_bse - iy) * (iz_bse - iz);
                            scalar_t tne = (ix - ix_bsw) * (iy_bsw - iy) * (iz_bsw - iz);
                            scalar_t tsw = (ix_bne - ix) * (iy - iy_bne) * (iz_bne - iz);
                            scalar_t tse = (ix - ix_bnw) * (iy - iy_bnw) * (iz_bnw - iz);
                            scalar_t bnw = (ix_tse - ix) * (iy_tse - iy) * (iz - iz_tse);
                            scalar_t bne = (ix - ix_tsw) * (iy_tsw - iy) * (iz - iz_tsw);
                            scalar_t bsw = (ix_tne - ix) * (iy - iy_tne) * (iz - iz_tne);
                            scalar_t bse = (ix - ix_tnw) * (iy - iy_tnw) * (iz - iz_tnw);

                            //We are reading from the same volume, no matter which batch
                            auto inp_ptr_NC = input.data;
                            auto out_ptr_NCHW = output.data + n * out_sN + h * out_sH + w * out_sW;

                            for (index_t c = 0; c < C; ++c, inp_ptr_NC += inp_sC, out_ptr_NCHW += out_sC) {
                                //   (c, iz_tnw, iy_tnw, ix_tnw) * tnw + (c, iz_tne, iy_tne, ix_tne) * tne
                                // + (c, iz_tsw, iy_tsw, ix_tsw) * tsw + (c, iz_tse, iy_tse, ix_tse) * tse
                                // + (c, iz_bnw, iy_bnw, ix_bnw) * bnw + (c, iz_bne, iy_bne, ix_bne) * bne
                                // + (c, iz_bsw, iy_bsw, ix_bsw) * bsw + (c, iz_bse, iy_bse, ix_bse) * bse
                                //*out_ptr_NCHW = static_cast<scalar_t>(0);
                                if (within_bounds_3d(iz_tnw, iy_tnw, ix_tnw, inp_D, inp_H, inp_W)) {
                                    *out_ptr_NCHW += inp_ptr_NC[iz_tnw * inp_sD + iy_tnw * inp_sH + ix_tnw * inp_sW] * tnw;
                                }
                                if (within_bounds_3d(iz_tne, iy_tne, ix_tne, inp_D, inp_H, inp_W)) {
                                    *out_ptr_NCHW += inp_ptr_NC[iz_tne * inp_sD + iy_tne * inp_sH + ix_tne * inp_sW] * tne;
                                }
                                if (within_bounds_3d(iz_tsw, iy_tsw, ix_tsw, inp_D, inp_H, inp_W)) {
                                    *out_ptr_NCHW += inp_ptr_NC[iz_tsw * inp_sD + iy_tsw * inp_sH + ix_tsw * inp_sW] * tsw;
                                }
                                if (within_bounds_3d(iz_tse, iy_tse, ix_tse, inp_D, inp_H, inp_W)) {
                                    *out_ptr_NCHW += inp_ptr_NC[iz_tse * inp_sD + iy_tse * inp_sH + ix_tse * inp_sW] * tse;
                                }
                                if (within_bounds_3d(iz_bnw, iy_bnw, ix_bnw, inp_D, inp_H, inp_W)) {
                                    *out_ptr_NCHW += inp_ptr_NC[iz_bnw * inp_sD + iy_bnw * inp_sH + ix_bnw * inp_sW] * bnw;
                                }
                                if (within_bounds_3d(iz_bne, iy_bne, ix_bne, inp_D, inp_H, inp_W)) {
                                    *out_ptr_NCHW += inp_ptr_NC[iz_bne * inp_sD + iy_bne * inp_sH + ix_bne * inp_sW] * bne;
                                }
                                if (within_bounds_3d(iz_bsw, iy_bsw, ix_bsw, inp_D, inp_H, inp_W)) {
                                    *out_ptr_NCHW += inp_ptr_NC[iz_bsw * inp_sD + iy_bsw * inp_sH + ix_bsw * inp_sW] * bsw;
                                }
                                if (within_bounds_3d(iz_bse, iy_bse, ix_bse, inp_D, inp_H, inp_W)) {
                                    *out_ptr_NCHW += inp_ptr_NC[iz_bse * inp_sD + iy_bse * inp_sH + ix_bse * inp_sW] * bse;

                                }

                            }
                        }
                    }

                }
            }

            template <typename scalar_t, typename index_t>
            __global__ void grid_sampler_and_project_3d_backward_kernel(
                    const index_t nthreads,
                    TensorInfo<scalar_t, index_t> grad_output,
                    TensorInfo<scalar_t, index_t> input,
                    TensorInfo<scalar_t, index_t> grid,
                    TensorInfo<scalar_t, index_t> grad_input,  // initialized to zeros
                    TensorInfo<scalar_t, index_t> grad_grid,   // initialized to empty
                    const GridSamplerInterpolation interpolation_mode,
                    const GridSamplerPadding padding_mode,
                    bool align_corners) {

                index_t C = input.sizes[1];
                index_t inp_D = input.sizes[2];
                index_t inp_H = input.sizes[3];
                index_t inp_W = input.sizes[4];
                index_t grid_D = grid.sizes[1];
                index_t out_H = grid.sizes[2];
                index_t out_W = grid.sizes[3];
                index_t inp_sN = input.strides[0];
                index_t inp_sC = input.strides[1];
                index_t inp_sD = input.strides[2];
                index_t inp_sH = input.strides[3];
                index_t inp_sW = input.strides[4];
                index_t grid_sN = grid.strides[0];
                index_t grid_sD = grid.strides[1];
                index_t grid_sH = grid.strides[2];
                index_t grid_sW = grid.strides[3];
                index_t grid_sCoor = grid.strides[4];
                index_t gOut_sN = grad_output.strides[0];
                index_t gOut_sC = grad_output.strides[1];
                //index_t gOut_sD = grad_output.strides[2];
                index_t gOut_sH = grad_output.strides[2];
                index_t gOut_sW = grad_output.strides[3];
                //index_t gInp_sN = grad_input.strides[0];
                index_t gInp_sC = grad_input.strides[1];
                index_t gInp_sD = grad_input.strides[2];
                index_t gInp_sH = grad_input.strides[3];
                index_t gInp_sW = grad_input.strides[4];
                index_t gGrid_sW = grad_grid.strides[3];

                CUDA_KERNEL_LOOP_TYPE(index, nthreads, index_t) {
                    const index_t w = index % out_W;
                    const index_t h = (index / out_W) % out_H;
                    //const index_t d = (index / (out_H * out_W)) % grid_D;
                    const index_t n = index / (out_H * out_W);
                    for (index_t d = 0; d < grid_D; d++) {
                        const auto grid_offset = n * grid_sN + d * grid_sD + h * grid_sH + w * grid_sW;

                        // get the corresponding input x, y, z co-ordinates from grid
                        scalar_t ix = grid.data[grid_offset];
                        scalar_t iy = grid.data[grid_offset + grid_sCoor];
                        scalar_t iz = grid.data[grid_offset + 2 * grid_sCoor];

                        // multipliers for gradients on ix, iy, and iz
                        scalar_t gix_mult, giy_mult, giz_mult;
                        ix = atoms_to_grid_compute_source_index_set_grad(ix, inp_W, padding_mode, align_corners, &gix_mult);
                        iy = atoms_to_grid_compute_source_index_set_grad(iy, inp_H, padding_mode, align_corners, &giy_mult);
                        iz = atoms_to_grid_compute_source_index_set_grad(iz, inp_D, padding_mode, align_corners, &giz_mult);

                        if (interpolation_mode == GridSamplerInterpolation::Bilinear) {
                            // get corner pixel values from (x, y, z)
                            // for 4d, we used north-east-south-west
                            // for 5d, we add top-bottom
                            index_t ix_tnw = static_cast<index_t>(::floor(ix));
                            index_t iy_tnw = static_cast<index_t>(::floor(iy));
                            index_t iz_tnw = static_cast<index_t>(::floor(iz));

                            index_t ix_tne = ix_tnw + 1;
                            index_t iy_tne = iy_tnw;
                            index_t iz_tne = iz_tnw;

                            index_t ix_tsw = ix_tnw;
                            index_t iy_tsw = iy_tnw + 1;
                            index_t iz_tsw = iz_tnw;

                            index_t ix_tse = ix_tnw + 1;
                            index_t iy_tse = iy_tnw + 1;
                            index_t iz_tse = iz_tnw;

                            index_t ix_bnw = ix_tnw;
                            index_t iy_bnw = iy_tnw;
                            index_t iz_bnw = iz_tnw + 1;

                            index_t ix_bne = ix_tnw + 1;
                            index_t iy_bne = iy_tnw;
                            index_t iz_bne = iz_tnw + 1;

                            index_t ix_bsw = ix_tnw;
                            index_t iy_bsw = iy_tnw + 1;
                            index_t iz_bsw = iz_tnw + 1;

                            index_t ix_bse = ix_tnw + 1;
                            index_t iy_bse = iy_tnw + 1;
                            index_t iz_bse = iz_tnw + 1;

                            // get surfaces to each neighbor:
                            scalar_t tnw = (ix_bse - ix) * (iy_bse - iy) * (iz_bse - iz);
                            scalar_t tne = (ix - ix_bsw) * (iy_bsw - iy) * (iz_bsw - iz);
                            scalar_t tsw = (ix_bne - ix) * (iy - iy_bne) * (iz_bne - iz);
                            scalar_t tse = (ix - ix_bnw) * (iy - iy_bnw) * (iz_bnw - iz);
                            scalar_t bnw = (ix_tse - ix) * (iy_tse - iy) * (iz - iz_tse);
                            scalar_t bne = (ix - ix_tsw) * (iy_tsw - iy) * (iz - iz_tsw);
                            scalar_t bsw = (ix_tne - ix) * (iy - iy_tne) * (iz - iz_tne);
                            scalar_t bse = (ix - ix_tnw) * (iy - iy_tnw) * (iz - iz_tnw);

                            scalar_t gix = static_cast<scalar_t>(0), giy = static_cast<scalar_t>(0), giz = static_cast<scalar_t>(0);
                            scalar_t* gOut_ptr_NCHW = grad_output.data + n * gOut_sN + h * gOut_sH + w * gOut_sW;
                            // Always use the same volume to reduce memory imprint
                            scalar_t* gInp_ptr_NC = grad_input.data;
                            scalar_t* inp_ptr_NC = input.data;
                            // calculate bilinear weighted pixel value and set output pixel
                            for (index_t c = 0; c < C; ++c, gOut_ptr_NCHW += gOut_sC, gInp_ptr_NC += gInp_sC, inp_ptr_NC += inp_sC) {
                                scalar_t gOut = *gOut_ptr_NCHW;

                                // calculate and set grad_input
                                safe_add_3d(gInp_ptr_NC, iz_tnw, iy_tnw, ix_tnw, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, tnw * gOut);
                                safe_add_3d(gInp_ptr_NC, iz_tne, iy_tne, ix_tne, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, tne * gOut);
                                safe_add_3d(gInp_ptr_NC, iz_tsw, iy_tsw, ix_tsw, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, tsw * gOut);
                                safe_add_3d(gInp_ptr_NC, iz_tse, iy_tse, ix_tse, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, tse * gOut);
                                safe_add_3d(gInp_ptr_NC, iz_bnw, iy_bnw, ix_bnw, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, bnw * gOut);
                                safe_add_3d(gInp_ptr_NC, iz_bne, iy_bne, ix_bne, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, bne * gOut);
                                safe_add_3d(gInp_ptr_NC, iz_bsw, iy_bsw, ix_bsw, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, bsw * gOut);
                                safe_add_3d(gInp_ptr_NC, iz_bse, iy_bse, ix_bse, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, bse * gOut);

                                // calculate grad_grid
                                if (within_bounds_3d(iz_tnw, iy_tnw, ix_tnw, inp_D, inp_H, inp_W)) {
                                    scalar_t tnw_val = inp_ptr_NC[iz_tnw * inp_sD + iy_tnw * inp_sH + ix_tnw * inp_sW];
                                    gix -= tnw_val * (iy_bse - iy) * (iz_bse - iz) * gOut;
                                    giy -= tnw_val * (ix_bse - ix) * (iz_bse - iz) * gOut;
                                    giz -= tnw_val * (ix_bse - ix) * (iy_bse - iy) * gOut;
                                }
                                if (within_bounds_3d(iz_tne, iy_tne, ix_tne, inp_D, inp_H, inp_W)) {
                                    scalar_t tne_val = inp_ptr_NC[iz_tne * inp_sD + iy_tne * inp_sH + ix_tne * inp_sW];
                                    gix += tne_val * (iy_bsw - iy) * (iz_bsw - iz) * gOut;
                                    giy -= tne_val * (ix - ix_bsw) * (iz_bsw - iz) * gOut;
                                    giz -= tne_val * (ix - ix_bsw) * (iy_bsw - iy) * gOut;
                                }
                                if (within_bounds_3d(iz_tsw, iy_tsw, ix_tsw, inp_D, inp_H, inp_W)) {
                                    scalar_t tsw_val = inp_ptr_NC[iz_tsw * inp_sD + iy_tsw * inp_sH + ix_tsw * inp_sW];
                                    gix -= tsw_val * (iy - iy_bne) * (iz_bne - iz) * gOut;
                                    giy += tsw_val * (ix_bne - ix) * (iz_bne - iz) * gOut;
                                    giz -= tsw_val * (ix_bne - ix) * (iy - iy_bne) * gOut;
                                }
                                if (within_bounds_3d(iz_tse, iy_tse, ix_tse, inp_D, inp_H, inp_W)) {
                                    scalar_t tse_val = inp_ptr_NC[iz_tse * inp_sD + iy_tse * inp_sH + ix_tse * inp_sW];
                                    gix += tse_val * (iy - iy_bnw) * (iz_bnw - iz) * gOut;
                                    giy += tse_val * (ix - ix_bnw) * (iz_bnw - iz) * gOut;
                                    giz -= tse_val * (ix - ix_bnw) * (iy - iy_bnw) * gOut;
                                }
                                if (within_bounds_3d(iz_bnw, iy_bnw, ix_bnw, inp_D, inp_H, inp_W)) {
                                    scalar_t bnw_val = inp_ptr_NC[iz_bnw * inp_sD + iy_bnw * inp_sH + ix_bnw * inp_sW];
                                    gix -= bnw_val * (iy_tse - iy) * (iz - iz_tse) * gOut;
                                    giy -= bnw_val * (ix_tse - ix) * (iz - iz_tse) * gOut;
                                    giz += bnw_val * (ix_tse - ix) * (iy_tse - iy) * gOut;
                                }
                                if (within_bounds_3d(iz_bne, iy_bne, ix_bne, inp_D, inp_H, inp_W)) {
                                    scalar_t bne_val = inp_ptr_NC[iz_bne * inp_sD + iy_bne * inp_sH + ix_bne * inp_sW];
                                    gix += bne_val * (iy_tsw - iy) * (iz - iz_tsw) * gOut;
                                    giy -= bne_val * (ix - ix_tsw) * (iz - iz_tsw) * gOut;
                                    giz += bne_val * (ix - ix_tsw) * (iy_tsw - iy) * gOut;
                                }
                                if (within_bounds_3d(iz_bsw, iy_bsw, ix_bsw, inp_D, inp_H, inp_W)) {
                                    scalar_t bsw_val = inp_ptr_NC[iz_bsw * inp_sD + iy_bsw * inp_sH + ix_bsw * inp_sW];
                                    gix -= bsw_val * (iy - iy_tne) * (iz - iz_tne) * gOut;
                                    giy += bsw_val * (ix_tne - ix) * (iz - iz_tne) * gOut;
                                    giz += bsw_val * (ix_tne - ix) * (iy - iy_tne) * gOut;
                                }
                                if (within_bounds_3d(iz_bse, iy_bse, ix_bse, inp_D, inp_H, inp_W)) {
                                    scalar_t bse_val = inp_ptr_NC[iz_bse * inp_sD + iy_bse * inp_sH + ix_bse * inp_sW];
                                    gix += bse_val * (iy - iy_tnw) * (iz - iz_tnw) * gOut;
                                    giy += bse_val * (ix - ix_tnw) * (iz - iz_tnw) * gOut;
                                    giz += bse_val * (ix - ix_tnw) * (iy - iy_tnw) * gOut;
                                }
                            }

                            // assuming grad_grid is contiguous
                            // thus we can
                            //   1. use index with gGrid_sW to directly compute gGrid_ptr_NDHW
                            //   2. directly assign to gGrid_ptr_NDHW[0], gGrid_ptr_NDHW[1], gGrid_ptr_NDHW[2]
                            scalar_t* gGrid_ptr_NDHW = grad_grid.data + n*grid_sN + d*grid_sD + h*grid_sH + w * gGrid_sW;
                            gGrid_ptr_NDHW[0] = gix_mult * gix;
                            gGrid_ptr_NDHW[1] = giy_mult * giy;
                            gGrid_ptr_NDHW[2] = giz_mult * giz;
                        }

                    }
                }
            }
            
            // No shape checking needed here. See # NOTE [ atoms_to_grid Native Functions ].
            Tensor atoms_to_grid_3d_cuda(const Tensor& intensities, const Tensor& positions, const Tensor& orientations, const Tensor& shift, int64_t x, int64_t y, int64_t z) {
                auto N = positions.size(0);
                auto W = positions.size(1);
                auto output = at::zeros({ N, z, y, x }, intensities.options());
                int64_t count = N * W;
                GridSamplerInterpolation interpolation_mode = GridSamplerInterpolation::Bilinear;
                GridSamplerPadding padding_mode = GridSamplerPadding::Zeros;
                bool align_corners = true;
                if (count > 0) {
                    AT_DISPATCH_FLOATING_TYPES_AND_HALF(intensities.scalar_type(), "atoms_to_grid_3d_cuda", [&] {
                        if (canUse32BitIndexMath(intensities) && canUse32BitIndexMath(positions) &&
                            canUse32BitIndexMath(output)) {
                            atoms_to_grid_3d_kernel<scalar_t>
                                << <MY_CUDA_GET_BLOCKS(count), MY_CUDA_MAX_THREADS, 0, at::cuda::getCurrentCUDAStream() >> > (
                                    static_cast<int>(count),
                                    getTensorInfo<scalar_t, int>(intensities),
                                    getTensorInfo<scalar_t, int>(positions),
                                    getTensorInfo<scalar_t, int>(orientations),
                                    getTensorInfo<scalar_t, int>(shift),
                                    getTensorInfo<scalar_t, int>(output),
                                    static_cast<GridSamplerInterpolation>(interpolation_mode),
                                    static_cast<GridSamplerPadding>(padding_mode),
                                    align_corners);
                            AT_CUDA_CHECK(hipGetLastError());
                        }
                        else {
                            atoms_to_grid_3d_kernel<scalar_t>
                                << <MY_CUDA_GET_BLOCKS(count), MY_CUDA_MAX_THREADS, 0, at::cuda::getCurrentCUDAStream() >> > (
                                    count,
                                    getTensorInfo<scalar_t, int64_t>(intensities),
                                    getTensorInfo<scalar_t, int64_t>(positions),
                                    getTensorInfo<scalar_t, int64_t>(orientations),
                                    getTensorInfo<scalar_t, int64_t>(shift),
                                    getTensorInfo<scalar_t, int64_t>(output),
                                    static_cast<GridSamplerInterpolation>(interpolation_mode),
                                    static_cast<GridSamplerPadding>(padding_mode),
                                    align_corners);
                            AT_CUDA_CHECK(hipGetLastError());
                        }
                        });
                }
                return output;
            }

            // No shape checking needed here. See # NOTE [ atoms_to_grid Native Functions ].
            std::tuple<Tensor, Tensor, Tensor, Tensor>
                atoms_to_grid_3d_backward_cuda(const Tensor& grad_output, const Tensor& intensities,
                    const Tensor& positions, const Tensor& orientations, const Tensor& shift) {
                // See Note [Writing Nondeterministic Operations]
                // Nondeterministic because of atomicAdd usage
                globalContext().alertNotDeterministic("atoms_to_grid_3d_backward_cuda");
                auto N = positions.size(0);
                auto W = positions.size(1);
                int64_t count = N * W;
                GridSamplerInterpolation interpolation_mode = GridSamplerInterpolation::Bilinear;
                GridSamplerPadding padding_mode = GridSamplerPadding::Zeros;
                bool align_corners = true;
                auto grad_intensities = at::zeros_like(intensities, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
                auto grad_positions = at::empty_like(positions, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
                auto grad_orientations = at::zeros_like(orientations, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
                if (count > 0) {
                    AT_DISPATCH_FLOATING_TYPES_AND_HALF(intensities.scalar_type(), "atoms_to_grid_3d_backward_cuda", [&] {

                        if (canUse32BitIndexMath(intensities) && canUse32BitIndexMath(positions) &&
                            canUse32BitIndexMath(grad_output)) {

                            atoms_to_grid_3d_backward_kernel<scalar_t>
                                << <MY_CUDA_GET_BLOCKS(count), MY_CUDA_MAX_THREADS, 0, at::cuda::getCurrentCUDAStream() >> > (
                                    static_cast<int>(count),
                                    getTensorInfo<scalar_t, int>(grad_output),
                                    getTensorInfo<scalar_t, int>(intensities),
                                    getTensorInfo<scalar_t, int>(positions),
                                    getTensorInfo<scalar_t, int>(orientations),
                                    getTensorInfo<scalar_t, int>(shift),
                                    getTensorInfo<scalar_t, int>(grad_intensities),
                                    getTensorInfo<scalar_t, int>(grad_positions),
                                    getTensorInfo<scalar_t, int>(grad_orientations),
                                    static_cast<GridSamplerInterpolation>(interpolation_mode),
                                    static_cast<GridSamplerPadding>(padding_mode),
                                    align_corners);
                            AT_CUDA_CHECK(hipGetLastError());
                        }
                        else {
                            atoms_to_grid_3d_backward_kernel<scalar_t>
                                << <MY_CUDA_GET_BLOCKS(count), MY_CUDA_MAX_THREADS, 0, at::cuda::getCurrentCUDAStream() >> > (
                                    count,
                                    getTensorInfo<scalar_t, int64_t>(grad_output),
                                    getTensorInfo<scalar_t, int64_t>(intensities),
                                    getTensorInfo<scalar_t, int64_t>(positions),
                                    getTensorInfo<scalar_t, int64_t>(orientations),
                                    getTensorInfo<scalar_t, int64_t>(shift),
                                    getTensorInfo<scalar_t, int64_t>(grad_intensities),
                                    getTensorInfo<scalar_t, int64_t>(grad_positions),
                                    getTensorInfo<scalar_t, int64_t>(grad_orientations),
                                    static_cast<GridSamplerInterpolation>(interpolation_mode),
                                    static_cast<GridSamplerPadding>(padding_mode),
                                    align_corners);
                            AT_CUDA_CHECK(hipGetLastError());
                        }
                        });
                }
                auto grad_shift = grad_positions.sum(1);
                return std::make_tuple(grad_intensities, grad_positions, grad_orientations, grad_shift);
            }


            Tensor projectAtoms(const Tensor& intensities, const Tensor& positions, const Tensor& orientation, const Tensor& shift, int64_t x, int64_t y, int64_t z) {
                auto N = positions.size(0);
                auto W = positions.size(1);

                auto output = at::zeros({ N, y, x}, positions.options());
                int64_t count = N*W;
                bool align_corners = true;
                if (count > 0) {
                    AT_DISPATCH_FLOATING_TYPES_AND_HALF(intensities.scalar_type(), "projectAtoms", [&] {

                        if (canUse32BitIndexMath(positions) &&
                            canUse32BitIndexMath(intensities) && canUse32BitIndexMath(orientation) &&
                            canUse32BitIndexMath(output)) {
                            projectAtoms_kernel<scalar_t>
                                << <MY_CUDA_GET_BLOCKS(count), MY_CUDA_MAX_THREADS, 0, at::cuda::getCurrentCUDAStream() >> > (
                                    static_cast<int>(count),
                                    getTensorInfo<scalar_t, int>(positions),
                                    getTensorInfo<scalar_t, int>(intensities),
                                    getTensorInfo<scalar_t, int>(orientation),
                                    getTensorInfo<scalar_t, int>(shift),
                                    getTensorInfo<scalar_t, int>(output), x, y, z);
                            AT_CUDA_CHECK(hipGetLastError());
                        }
                        else {
                            projectAtoms_kernel<scalar_t>
                                << <MY_CUDA_GET_BLOCKS(count), MY_CUDA_MAX_THREADS, 0, at::cuda::getCurrentCUDAStream() >> > (
                                    count,
                                    getTensorInfo<scalar_t, int64_t>(positions),
                                    getTensorInfo<scalar_t, int64_t>(intensities),
                                    getTensorInfo<scalar_t, int64_t>(orientation),
                                    getTensorInfo<scalar_t, int64_t>(shift),
                                    getTensorInfo<scalar_t, int64_t>(output), x, y, z);
                            AT_CUDA_CHECK(hipGetLastError());
                        }
                        });
                }
                return output;// .sum(1, false);
            }
            
            std::tuple<Tensor, Tensor, Tensor, Tensor>
                projectAtoms_backward_cuda(const Tensor& grad_output, const Tensor& intensities, const Tensor& positions, const Tensor& orientation, const Tensor& shift, int64_t x, int64_t y, int64_t z) {
                auto N = positions.size(0);
                auto W = positions.size(1);
                int64_t count = N * W;
                bool align_corners = true;
                auto grad_intensities = at::zeros_like(intensities, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
                auto grad_positions = at::empty_like(positions, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
                auto grad_orientation = at::zeros_like(orientation, LEGACY_CONTIGUOUS_MEMORY_FORMAT);

                if (count > 0) {
                    AT_DISPATCH_FLOATING_TYPES_AND_HALF(intensities.scalar_type(), "projectAtoms", [&] {
                        if (canUse32BitIndexMath(positions) &&
                            canUse32BitIndexMath(intensities) && canUse32BitIndexMath(orientation) &&
                            canUse32BitIndexMath(grad_output)) {
                            projectAtoms_backwards_kernel<scalar_t>
                                << <MY_CUDA_GET_BLOCKS(count), MY_CUDA_MAX_THREADS, 0, at::cuda::getCurrentCUDAStream() >> > (
                                    static_cast<int>(count),
                                    getTensorInfo<scalar_t, int>(positions),
                                    getTensorInfo<scalar_t, int>(intensities),
                                    getTensorInfo<scalar_t, int>(orientation),
                                    getTensorInfo<scalar_t, int>(shift),
                                    getTensorInfo<scalar_t, int>(grad_output),
                                    getTensorInfo<scalar_t, int>(grad_positions),
                                    getTensorInfo<scalar_t, int>(grad_intensities),
                                    getTensorInfo<scalar_t, int>(grad_orientation), x, y, z);
                            AT_CUDA_CHECK(hipGetLastError());
                        }
                        else {

                            projectAtoms_backwards_kernel<scalar_t>
                                << <MY_CUDA_GET_BLOCKS(count), MY_CUDA_MAX_THREADS, 0, at::cuda::getCurrentCUDAStream() >> > (
                                    count,
                                    getTensorInfo<scalar_t, int64_t>(positions),
                                    getTensorInfo<scalar_t, int64_t>(intensities),
                                    getTensorInfo<scalar_t, int64_t>(orientation),
                                    getTensorInfo<scalar_t, int64_t>(shift),
                                    getTensorInfo<scalar_t, int64_t>(grad_output),
                                    getTensorInfo<scalar_t, int64_t>(grad_positions),
                                    getTensorInfo<scalar_t, int64_t>(grad_intensities),
                                    getTensorInfo<scalar_t, int64_t>(grad_orientation), x, y, z);
                            AT_CUDA_CHECK(hipGetLastError());
                        }
                        });
                }
                //Lazy Way: Shift gradient is just sum over all atoms. In the kernel, we would need atomic Add operation for this
                auto grad_shift = grad_positions.sum(1);
                return std::make_tuple(grad_intensities, grad_positions, grad_orientation, grad_shift);  
            }

            // No shape checking needed here. See # NOTE [ grid_sampler Native Functions ].
            Tensor grid_sampler_and_project_3d_cuda(const Tensor& input, const Tensor& grid,
                int64_t interpolation_mode, int64_t padding_mode,
                bool align_corners) {
                auto N = grid.size(0);
                //auto D = grid.size(1);
                auto H = grid.size(2);
                auto W = grid.size(3);
                auto output = at::zeros({ N, input.size(1), H, W }, input.options());
                int64_t count = N * H * W;

                auto inpDims = input.sizes().vec();
                auto gridDims = grid.sizes().vec();
                auto outDim = output.sizes().vec();

                if (count > 0) {
                    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "grid_sampler_and_project_3d_cuda", [&] {
                        if (canUse32BitIndexMath(input) && canUse32BitIndexMath(grid) &&
                            canUse32BitIndexMath(output)) {
                            grid_sampler_and_project_3d_kernel<scalar_t>
                                << <MY_CUDA_GET_BLOCKS(count), MY_CUDA_MAX_THREADS, 0, at::cuda::getCurrentCUDAStream() >> > (
                                    static_cast<int>(count),
                                    getTensorInfo<scalar_t, int>(input),
                                    getTensorInfo<scalar_t, int>(grid),
                                    getTensorInfo<scalar_t, int>(output),
                                    static_cast<GridSamplerInterpolation>(interpolation_mode),
                                    static_cast<GridSamplerPadding>(padding_mode),
                                    align_corners);
                        }
                        else {
                            grid_sampler_and_project_3d_kernel<scalar_t>
                                << <1, 1, 0, at::cuda::getCurrentCUDAStream() >> > (
                                    count,
                                    getTensorInfo<scalar_t, int64_t>(input),
                                    getTensorInfo<scalar_t, int64_t>(grid),
                                    getTensorInfo<scalar_t, int64_t>(output),
                                    static_cast<GridSamplerInterpolation>(interpolation_mode),
                                    static_cast<GridSamplerPadding>(padding_mode),
                                    align_corners);
                        }
                        });
                }
                return output;
            }

            // No shape checking needed here. See # NOTE [ grid_sampler Native Functions ].
            std::tuple<Tensor, Tensor>
                grid_sampler_and_project_3d_backward_cuda(const Tensor& grad_output, const Tensor& input,
                    const Tensor& grid, int64_t interpolation_mode, int64_t padding_mode,
                    bool align_corners) {
                // See Note [Writing Nondeterministic Operations]
                // Nondeterministic because of atomicAdd usage
                globalContext().alertNotDeterministic("grid_sampler_3d_backward_cuda");
                auto N = input.size(0);
                auto D = grid.size(1);
                auto H = grid.size(2);
                auto W = grid.size(3);
                auto grad_input = at::zeros_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
                auto grad_grid = at::empty_like(grid, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
                int64_t count = N * H * W;
                if (count > 0) {
                    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "grid_sampler_and_project_3d_backward_cuda", [&] {
                        if (canUse32BitIndexMath(input) && canUse32BitIndexMath(grid) &&
                            canUse32BitIndexMath(grad_output)) {
                            grid_sampler_and_project_3d_backward_kernel<scalar_t>
                                << <MY_CUDA_GET_BLOCKS(count), MY_CUDA_MAX_THREADS, 0, at::cuda::getCurrentCUDAStream() >> > (
                                    static_cast<int>(count),
                                    getTensorInfo<scalar_t, int>(grad_output),
                                    getTensorInfo<scalar_t, int>(input),
                                    getTensorInfo<scalar_t, int>(grid),
                                    getTensorInfo<scalar_t, int>(grad_input),
                                    getTensorInfo<scalar_t, int>(grad_grid),
                                    static_cast<GridSamplerInterpolation>(interpolation_mode),
                                    static_cast<GridSamplerPadding>(padding_mode),
                                    align_corners);
                        }
                        else {
                            grid_sampler_and_project_3d_backward_kernel<scalar_t>
                                << <MY_CUDA_GET_BLOCKS(count), MY_CUDA_MAX_THREADS, 0, at::cuda::getCurrentCUDAStream() >> > (
                                    count,
                                    getTensorInfo<scalar_t, int64_t>(grad_output),
                                    getTensorInfo<scalar_t, int64_t>(input),
                                    getTensorInfo<scalar_t, int64_t>(grid),
                                    getTensorInfo<scalar_t, int64_t>(grad_input),
                                    getTensorInfo<scalar_t, int64_t>(grad_grid),
                                    static_cast<GridSamplerInterpolation>(interpolation_mode),
                                    static_cast<GridSamplerPadding>(padding_mode),
                                    align_corners);
                        }
                        });
                }
                return std::make_tuple(grad_input, grad_grid);
            }

        }  // namespace
    }
}
    
